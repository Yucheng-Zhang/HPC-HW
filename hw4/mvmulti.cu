
#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void vvmulti(double* sum_ptr, const double* a, const double* b, long N){
  double sum = 0;
  #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < N; i++) sum += a[i] * b[i];
  *sum_ptr = sum;
}

#define BLOCK_SIZE 1024

__global__ void vvmulti_kernel(double* sum, const double* a, const double* b, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx] * b[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void reduction_kernel(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

int main() {
  long N = (1UL<<25);

  /* Initialize two vectors */
  double *a, *b;
  hipHostMalloc((void**)&a, N*sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&b, N*sizeof(double), hipHostMallocDefault);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    a[i] = 1.0 / (i+1);
    b[i] = 1.0;
  }

  /* CPU reference */
  double sum_ref;
  double tt = omp_get_wtime();
  vvmulti(&sum_ref, a, b, N);
  printf("CPU Bandwidth = %f GB/s\n", 2*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  /* GPU */
  double *a_d, *b_d, *sum_d;
  hipMalloc(&a_d, N*sizeof(double));
  hipMalloc(&b_d, N*sizeof(double));
  hipMalloc(&sum_d, ((N+BLOCK_SIZE-1)/BLOCK_SIZE)*sizeof(double));
  
  /* Copy Host data to device */
  hipMemcpyAsync(a_d, a, N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  hipMemcpyAsync(b_d, b, N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  tt = omp_get_wtime();

  long Nb = (N+BLOCK_SIZE-1)/BLOCK_SIZE;
  vvmulti_kernel<<<Nb,BLOCK_SIZE>>>(sum_d, a_d, b_d, N);
  while (Nb > 1) {
    long N = Nb;
    Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
    reduction_kernel<<<Nb,BLOCK_SIZE>>>(sum_d+Nb, sum_d, N);
    sum_d += Nb;
  }

  double sum;
  hipMemcpyAsync(&sum, sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  printf("GPU Bandwidth = %f GB/s\n", 2*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  printf("Error = %f\n", fabs(sum-sum_ref));

  hipFree(a_d);
  hipFree(b_d);
  hipFree(sum_d);
  hipHostFree(a);
  hipHostFree(b);

  return 0;
}
